#include "hip/hip_runtime.h"
#include "lab1.h"
static const unsigned W = 1366;
static const unsigned H = 768;
static const unsigned NFRAME = 240;

#define N_PARTICLES 10000
#define SCALE (1.0 / 100)
#define INIT_DISTANCE 500
#define GRAVITY 50
#define G_REPULSION 10.0

// typedef double2 Coord;
// typedef double2 Velocity;
// typedef short3* Velocity;

inline __host__ __device__ void operator*=(double2&a, double b) {
    a.x *= b;
    a.y *= b;
}

inline __host__ __device__ void operator+=(double2&a, double2 b) {
    a.x += b.x;
    a.y += b.y;
}


inline __host__ __device__  double2 operator-(double2 a, double2 b) {
    return make_double2(a.x - b.x, a.y - b.y);
}

inline __host__ __device__  double2 operator+(double2 a, double2 b) {
    return make_double2(a.x + b.x, a.y + b.y);
}

inline __host__ __device__  double2 operator*(double2 a, double b) {
    return make_double2(a.x * b, a.y * b);
}

inline __host__ __device__  double2 operator/(double2 a, double b) {
    return make_double2(a.x / b, a.y / b);
}


__device__ double norm(double2 v) {
    return sqrt(v.x * v.x + v.y * v.y);
}


struct Lab1VideoGenerator::Impl {
	int t = 0;
    double2* coordinate;
    double2* velocity;
    double2* prev_coordinate;
    double2* prev_velocity;
    short3* canvas;
};



__global__ void fillKernel(short3* canvas, float alpha) {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < W * H) {
        canvas[i] = make_short3(0, 0, 0);
    }
}
    
void fill(short3* canvas, float alpha=1.0){
    fillKernel<<<((W * H + 255)/ 256),256>>>(canvas, alpha); 
}

__global__ void rgb2yuvKernel(short3* canvas, uint8_t* yuv) {
    auto x = blockDim.x * blockIdx.x + threadIdx.x;
    auto y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x < W && y < H) {
        auto c = canvas[y * W + x];
        yuv[y * W + x] = 0.299*c.x + 0.587*c.y + 0.114*c.z;
        if (x % 2 == 0 && y % 2 == 0) {
            auto c2 = canvas[(y + 0) * W + x + 1];
            auto c3 = canvas[(y + 1) * W + x + 0];
            auto c4 = canvas[(y + 1) * W + x + 1];
            c.x = (c.x + c2.x + c3.x + c4.x) / 4;
            c.y = (c.y + c2.y + c3.y + c4.y) / 4;
            c.z = (c.z + c2.z + c3.z + c4.z) / 4;
            auto indU = W*H + y/2 * W/2 + x/2;
            auto indV = W*H + W*H/4 + y/2 * W/2 + x/2;
            yuv[indU] = -0.169*c.x - 0.331*c.y + 0.500*c.z + 128; 
            yuv[indV] = 0.500*c.x - 0.419*c.y - 0.081*c.z + 128; 
        }
    }    
}

void rgb2yuv(short3* canvas, uint8_t* yuv) {
    dim3 dimBlock(16, 16);
    dim3 dimGrid((W + 15)/16, (H + 15)/16);
    rgb2yuvKernel<<<dimGrid, dimBlock>>>(canvas, yuv);    
}

__device__ void drawDot(double2 coord, short3 color, short3* canvas) {
    int x = coord.x * SCALE + 20;
    int y = coord.y * SCALE;
    if ( x >= 0 && x < W - 1 && y >= 0 && y < H - 1) {
        canvas[(y + 0)*W + x + 0] = color;
        canvas[(y + 0)*W + x + 1] = color;
        canvas[(y + 1)*W + x + 0] = color;
        canvas[(y + 1)*W + x + 1] = color;
    }
}


__global__ void initParticlesKernel(double2* coord, double2* velocity, short3* canvas) {
    auto i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N_PARTICLES) {
        int x = i % 100;
        int y = i / 100;
        coord[i] = make_double2(x * INIT_DISTANCE, y * INIT_DISTANCE);
        velocity[i] = make_double2(0, 0);
    }
}


// __device__ void cohesion(


void initParticles(double2* coord, double2* velocity, short3* canvas) {
    initParticlesKernel<<<(N_PARTICLES + 255) / 256, 256>>>(coord, velocity, canvas);
}


__global__ void updateParticlesKernel(double2* prev_coord, double2* prev_velocity,
                                      double2* coord, double2* velocity, short3* canvas) {
    const short3 WHITE = make_short3(255, 255, 255);

    auto i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N_PARTICLES) {
        velocity[i] = prev_velocity[i];
        coord[i] = prev_coord[i];

        // reflection
        if (coord[i].y >= (H - 10)/SCALE) {
            velocity[i].y = - velocity[i].y;
        }
        if (coord[i].x >= (W - 10)/SCALE) {
            velocity[i].x = - velocity[i].x;
        }
        if (coord[i].x <= 10/SCALE) {
            velocity[i].x = - velocity[i].x;
        }

        // calculate acceleration
        velocity[i].y += GRAVITY;
        for (int j = 0; j < N_PARTICLES; ++j) {
            auto d = coord[i] - coord[j];
            auto d2 = d.x * d.x + d.y * d.y;
            d = d * G_REPULSION / (d2 * sqrt(d2));
            velocity[i] += d;
        }
            
        
        // update coordinate
        coord[i] += velocity[i];
        
        // boundary
        coord[i].x = coord[i].x < 10/SCALE ? 10/SCALE : coord[i].x;
        coord[i].x = coord[i].x > (W - 10)/SCALE ? (W - 10)/SCALE : coord[i].x;
        coord[i].y = coord[i].y > (H - 10)/SCALE ? (H - 10)/SCALE : coord[i].y;
        
        drawDot(coord[i], WHITE, canvas);
    }
}


void updateParticles(double2* prev_coord, double2* prev_velocity,
                     double2* coord, double2* velocity, short3* canvas) {
    updateParticlesKernel<<<(N_PARTICLES + 31) / 32, 32>>>(prev_coord, prev_velocity,
                                                           coord, velocity, canvas);
}


Lab1VideoGenerator::Lab1VideoGenerator(): impl(new Impl) {
    hipMalloc(&(impl->velocity), sizeof(double2) * N_PARTICLES);
    hipMalloc(&(impl->coordinate), sizeof(double2) * N_PARTICLES);
    hipMalloc(&(impl->prev_velocity), sizeof(double2) * N_PARTICLES);
    hipMalloc(&(impl->prev_coordinate), sizeof(double2) * N_PARTICLES);
    hipMalloc(&(impl->canvas), sizeof(short3) * W * H * 3 / 2);
    fill(impl->canvas);
    initParticles(impl->coordinate, impl->velocity, impl->canvas);
}

Lab1VideoGenerator::~Lab1VideoGenerator() {}

void Lab1VideoGenerator::get_info(Lab1VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};


void Lab1VideoGenerator::Generate(uint8_t *yuv) {
	// hipMemset(yuv, (impl->t)*255/NFRAME, W*H);
	// hipMemset(yuv+W*H, 128, W*H/2);    
    for (int i = 0; i < 10; ++i) {
        fill(impl->canvas);
        hipMemcpy(impl->prev_coordinate, impl->coordinate,
                   sizeof(double2) * N_PARTICLES, hipMemcpyDeviceToDevice);
        hipMemcpy(impl->prev_velocity, impl->velocity,
                   sizeof(double2) * N_PARTICLES, hipMemcpyDeviceToDevice);
        updateParticles(impl->prev_coordinate, impl->prev_velocity,
                        impl->coordinate, impl->velocity, impl->canvas);    
        rgb2yuv(impl->canvas, yuv);
    }
    ++(impl->t);
}
