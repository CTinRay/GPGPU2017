#include "hip/hip_runtime.h"
#include "lab1.h"
static const unsigned W = 640;
static const unsigned H = 480;
static const unsigned NFRAME = 240;

#define N_PARTICLES 10000

struct Lab1VideoGenerator::Impl {
	int t = 0;
    double2* coordinate;
    double2* velocity;
    short3* canvas;
};



__global__ void fillKernel(short3* canvas, float alpha) {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < W * H) {
        canvas[i] = make_short3(0, 0, 0);
    }
}
    
void fill(short3* canvas, float alpha=1.0){
    fillKernel<<<((W * H + 255)/ 256),256>>>(canvas, alpha); 
}

__global__ void rgb2yuvKernel(short3* canvas, uint8_t* yuv) {
    auto x = blockDim.x * blockIdx.x + threadIdx.x;
    auto y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x < W && y < H) {
        auto c = canvas[y * W + x];
        yuv[y * W + x] = 0.299*c.x + 0.587*c.y + 0.114*c.z;
        if (x % 2 == 0 && y % 2 == 0) {
            auto c2 = canvas[(y + 0) * W + x + 1];
            auto c3 = canvas[(y + 1) * W + x + 0];
            auto c4 = canvas[(y + 1) * W + x + 1];
            c.x = (c.x + c2.x + c3.x + c4.x) / 4;
            c.y = (c.y + c2.y + c3.y + c4.y) / 4;
            c.z = (c.z + c2.z + c3.z + c4.z) / 4;
            auto indU = W*H + y/2 * W/2 + x/2;
            auto indV = W*H + W*H/4 + y/2 * W/2 + x/2;
            yuv[indU] = -0.169*c.x - 0.331*c.y + 0.500*c.z + 128; 
            yuv[indV] = 0.500*c.x - 0.419*c.y - 0.081*c.z + 128; 
        }
    }    
}

void rgb2yuv(short3* canvas, uint8_t* yuv) {
    dim3 dimBlock(16, 16);
    dim3 dimGrid((W + 15)/16, (H + 15)/16);
    rgb2yuvKernel<<<dimGrid, dimBlock>>>(canvas, yuv);    
}



Lab1VideoGenerator::Lab1VideoGenerator(): impl(new Impl) {
    hipMalloc(&(impl->velocity), sizeof(double2) * N_PARTICLES);
    hipMalloc(&(impl->coordinate), sizeof(double2) * N_PARTICLES);
    hipMalloc(&(impl->canvas), sizeof(short3) * W * H * 3 / 2);
    fill(impl->canvas);
}

Lab1VideoGenerator::~Lab1VideoGenerator() {}

void Lab1VideoGenerator::get_info(Lab1VideoInfo &info) {
	info.w = W;
	info.h = H;
	info.n_frame = NFRAME;
	// fps = 24/1 = 24
	info.fps_n = 24;
	info.fps_d = 1;
};


void Lab1VideoGenerator::Generate(uint8_t *yuv) {
	// hipMemset(yuv, (impl->t)*255/NFRAME, W*H);
	// hipMemset(yuv+W*H, 128, W*H/2);
    fill(impl->canvas);
    rgb2yuv(impl->canvas, yuv);
	++(impl->t);
}
